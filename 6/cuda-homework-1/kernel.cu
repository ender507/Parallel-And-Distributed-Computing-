#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <cmath>
#include "error_checks.h" // Macros CUDA_CHECK and CHECK_ERROR_MSG


__global__ void vector_add(double* C, const double* A, const double* B, int N)
{
	// Add the kernel code
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// Do not try to access past the allocated memory
	if (idx < N) {
		C[idx] = A[idx] + B[idx];
	}
}


int main(void)
{
	const int N = 20;
	const int ThreadsInBlock = 128;
	double* dA, * dB, * dC;
	double hA[N], hB[N], hC[N];

	for (int i = 0; i < N; ++i) {
		hA[i] = (double)i;
		hB[i] = (double)i * i;
	}

	/*
	   Add memory allocations and copies. Wrap your runtime function
	   calls with CUDA_CHECK( ) macro
	*/
	CUDA_CHECK(hipMalloc((void**)& dA, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((void**)& dB, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((void**)& dC, sizeof(double) * N));
	CUDA_CHECK(hipMemcpy(dA, hA, sizeof(double) * N, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(dB, hB, sizeof(double) * N, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(dC, hC, sizeof(double) * N, hipMemcpyHostToDevice));
	//#error Add the remaining memory allocations and copies
	// Note the maximum size of threads in a block
	dim3 grid, threads;

	// Add the kernel call here
//#error Add the CUDA kernel call
	vector_add << <1, ThreadsInBlock >> > (dC, dA, dB, N);

// Here we add an explicit synchronization so that we catch errors
// as early as possible. Don't do this in production code!
	hipDeviceSynchronize();
	CHECK_ERROR_MSG("vector_add kernel");

	// Copy back the results and free the device memory
//#error Copy back the results and free the allocated memory
	CUDA_CHECK(hipMemcpy(hA, dA, sizeof(double) * N, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(hB, dB, sizeof(double) * N, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(hC, dC, sizeof(double) * N, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipFree(dA));
	CUDA_CHECK(hipFree(dB));
	CUDA_CHECK(hipFree(dC));
	for (int i = 0; i < N; i++)
		printf("%5.1f\n", hC[i]);

	return 0;
}